#include <hip/hip_runtime.h>
#include <dlfcn.h>
#include <stdio.h>
//#include <cuda_runtime.h>
//#include <driver_types.h>

void cudaLaunchKernelHelper (hipStream_t hStream);

hipError_t hipLaunchKernel(hipFunction_t f, unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ, unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ, unsigned int sharedMemBytes, hipStream_t hStream, void** kernelParams, void** extra) {
	
	printf("hello!\n");

	void* handle;

	hipError_t (*function)(hipFunction_t f,  
                        unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ, 
                        unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ,
                        unsigned int sharedMemBytes, hipStream_t hStream, void** kernelParams, void** extra);


	*(void **)(&function) = dlsym (RTLD_NEXT, "hipLaunchKernel");

	cudaLaunchKernelHelper (hStream);

	(*function)(f, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, hStream, kernelParams, extra);

}

void cudaLaunchKernelHelper (hipStream_t hStream) {
	// Nothing
	printf ("cudaLaunchHelper\n");
}
