#include "intercept_temp.h"

using namespace std;

queue<kernel_record> kqueue0;
queue<kernel_record> kqueue1;
pthread_mutex_t mutex0;
pthread_mutex_t mutex1;
pthread_t thread_ids[2];

queue<kernel_record>* kqueues[2] = {&kqueue0, &kqueue1};
pthread_mutex_t* mutexes[2] = {&mutex0, &mutex1};
int i=0;

void print_kernel_invocation(int i, dim3 gridDim, dim3 blockDim) {
	
	printf("%d, ", i);
	if (gridDim.y == 1 && gridDim.z == 1) {
  		printf("--gridDim=%d ", gridDim.x);
	} else if (gridDim.z == 1) {
		printf("--gridDim=[%d,%d] ", gridDim.x, gridDim.y);
	} else {
		printf("--gridDim=[%d,%d,%d] ", gridDim.x, gridDim.y, gridDim.z);
	}

	if (blockDim.y == 1 && blockDim.z == 1) {
		printf("--blockDim=%d ", blockDim.x);
	} else if (blockDim.z == 1) {
		printf("--blockDim=[%d,%d] ", blockDim.x, blockDim.y);
	} else {
		printf("--blockDim=[%d,%d,%d] ", blockDim.x, blockDim.y, blockDim.z);
	}
	//printf("\n");
}


hipError_t hipMalloc(void** devPtr, size_t size) {

	printf("Caught hipMalloc! allocate region of %ld bytes\n", *devPtr, size);

	hipError_t (*function)(void** devPtr, size_t size);
	*(void **)(&function) = dlsym (RTLD_NEXT, "hipMalloc");
	
	hipError_t err = (*function)(devPtr, size);
	printf("Memory allocated at address %p\n", *devPtr);
	return err;

}


hipError_t hipFree(void* devPtr) {

	printf("Caught hipFree! Free pointer that holds address %p\n", devPtr);

	hipError_t (*function)(void* devPtr);
	*(void **)(&function) = dlsym (RTLD_NEXT, "hipFree");

	hipError_t err; //= (*function)(devPtr);
	return err;

}

hipError_t hipLaunchKernel(reinterpret_cast<const void*>( const void* func), dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream ) {


	printf("Captured a hipLaunchKernel! id is %d, function ptr is %p, stream is %d, gridDim is %d, blockDim is %d, sharedMem is %ld\n", i, func, stream, gridDim, blockDim, sharedMem);


	hipError_t (*function)(const void* func, dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream);
	*(void **)(&function) = dlsym (RTLD_NEXT, "hipLaunchKernel");
	hipError_t err = hipSuccess;

	struct kernel_record new_record = {func, gridDim, blockDim, args, sharedMem, stream, false, 0};

	// push at queue

#ifdef SYS_gettid
	pid_t tid = syscall(SYS_gettid);
#else
#error "SYS_gettid unavailable on this system"
#endif
	printf("My id is %d\n", tid);

	int idx=-1;
	if (tid == thread_ids[0])
		idx = 0;
	else if (tid == thread_ids[1])
		idx = 1;
	else
		idx = 0;
		//printf("----------------------- INVALID!!!!!!!!! -------------------\n");

	//printf("idx: %d, queues: %p, queue: %p, mutex: %p\n", idx, kqueues, kqueues[idx], mutexes[idx]);

	if (stream==0) {
	
		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);
	}
	else {
		printf("------------------------ before submitting\n");
		err = (*function)(func, gridDim, blockDim, args, sharedMem, stream);
		printf("------------------------ after submitting\n");
	}

	// wait and run
	/* while (true) {
		pthread_mutex_lock(mutexes[0]);
		if (kqueues[0]->front().run) {
			hipStream_t sched_stream = kqueues[0]->front().sched_stream;
			kqueues[0]->pop();
			printf("-------- run with stream %d!!!\n", sched_stream);
			pthread_mutex_unlock(mutexes[0]);
			err = (*function)(func, gridDim, blockDim, args, sharedMem, sched_stream); 
			return err;
		}
		pthread_mutex_unlock(mutexes[0]);
	} */
}

int main() {

	printf("running......\n");
}
