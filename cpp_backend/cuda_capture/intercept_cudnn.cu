
#include "intercept_temp.h"

hipdnnStatus_t hipdnnConvolutionForward(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) {


	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnConvolutionForward, CUDNN handle is %p, index is %d\n", func_indexes[idx], handle, idx);

	// create record
	cudnnConvolutionForward_record new_conv_record = {
		handle,
		alpha,
		xDesc,
		x,
		wDesc,
		w,
		convDesc,
		algo,
		workSpace,
		workSpaceSizeInBytes,
		beta,
		yDesc,
		y
	};
	union func_data new_func_data;
	new_func_data.cudnnConvRecord = new_conv_record;
	func_record new_record = {CUDNN_CONV_RECORD, new_func_data};



	// push or run
	if (idx < 2) {
		 pthread_mutex_lock(mutexes[idx]);
		 kqueues[idx]->push(new_record);
		 pthread_mutex_unlock(mutexes[idx]);

		 func_indexes[idx] += 1;
		 block(idx, mutexes, kqueues);
	}
	else {
		hipdnnStatus_t (*function)(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) ;
		*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnConvolutionForward");
		assert(function != NULL);

		status = (*function)(handle, alpha, xDesc, x, wDesc, w, convDesc, algo, workSpace, workSpaceSizeInBytes, beta, yDesc, y);
		assert (status == HIPDNN_STATUS_SUCCESS);

	}

	return status;

}

hipdnnStatus_t cudnnBatchNormalizationForwardTrainingEx(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught cudnnBatchNormalizationForwardTrainingEx, handle is %p, index is %d\n", func_indexes[idx], handle, idx);


	// create record
	cudnnBatchNormalizationForwardTrainingEx_record new_bn_record = {
		handle,
		mode,
		bnOps,
		alpha,
		beta,
		xDesc,
		xData,
		zDesc,
		zData,
		yDesc,
		yData,
		bnScaleBiasMeanVarDesc,
		bnScaleData,
		bnBiasData,
		exponentialAverageFactor,
		resultRunningMeanData,
		resultRunningVarianceData,
		epsilon,
		saveMean,
		saveInvVariance,
		activationDesc,
		workspace,
		workSpaceSizeInBytes,
		reserveSpace,
		reserveSpaceSizeInBytes

	};
	union func_data new_func_data;
	new_func_data.cudnnBNormRecord = new_bn_record;
	func_record new_record = {CUDNN_BNORM_RECORD, new_func_data};

	// push or run

	if (idx < 2) {
		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);
	}
	else {
		hipdnnStatus_t (*function)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes);

		*(void **)(&function) = dlsym(RTLD_NEXT, "cudnnBatchNormalizationForwardTrainingEx");
		assert(function != NULL);

		status = (*function)(handle, mode, bnOps, alpha, beta, xDesc, xData, zDesc, zData, yDesc, yData, bnScaleBiasMeanVarDesc, bnScaleData, bnBiasData, exponentialAverageFactor, resultRunningMeanData, resultRunningVarianceData, epsilon, saveMean, saveInvVariance, activationDesc, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);
		assert (status == HIPDNN_STATUS_SUCCESS);

	}

	return status;
}


hipdnnStatus_t hipdnnBatchNormalizationForwardInference(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon)

{

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnBatchNormalizationForwardInference, handle is %p, index is %d\n", func_indexes[idx], handle, idx);



	// create record
	cudnnBatchNormalizationForwardInference_record bn_record = {
		handle,
		mode,
		alpha,
		beta,
		xDesc,
		x,
		yDesc,
		y,
		bnScaleBiasMeanVarDesc,
		bnScale,
		bnBias,
		estimatedMean,
		estimatedVariance,
		epsilon
	};

	union func_data new_func_data;
	new_func_data.cudnnBNormInfRecord = bn_record;
	func_record new_record = {CUDNN_BNORM_INF_RECORD, new_func_data};

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);

	}
	else {

		hipdnnStatus_t (*function)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon);

		*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnBatchNormalizationForwardInference");
		assert(function != NULL);

		status = (*function)(handle, mode, alpha, beta, xDesc, x, xDesc, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, estimatedMean, estimatedVariance, epsilon);
		assert (status == HIPDNN_STATUS_SUCCESS);

	}

	return status;
}


hipdnnStatus_t hipdnnRNNForwardInference(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes)  {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;


	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnRNNForwardInference, handle is %p, index is %d\n", func_indexes[idx], handle, idx);
	printf("------------------------------------------------- IDX [%d], CX IS %p, CY IS %p\n", idx, cx, cy);

	if (idx < 2) {

		hipdnnTensorDescriptor_t* xDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
	        //hipdnnStatus_t s = hipdnnCreateTensorDescriptor(xDesc_new);

		*xDesc_new = *xDesc;
		printf("%p, %p, %p, %p\n", xDesc, *xDesc, xDesc_new, *(xDesc_new));
		//memcpy(xDesc_new, xDesc, sizeof(hipdnnTensorDescriptor_t));

		hipdnnTensorDescriptor_t* yDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
		*yDesc_new = *yDesc;


		cudnnRNNForwardInference_record rnn_record = {
			handle,
			rnnDesc,
			seqLength,
			xDesc_new,
			x,
			hxDesc,
			hx,
			cxDesc,
			cx,
			wDesc,
			w,
			yDesc_new,
			y,
			hyDesc,
			hy,
			cyDesc,
			cy,
			workspace,
			workSpaceSizeInBytes
		};

		union func_data new_func_data;
		new_func_data.cudnnRnnInfRecord = rnn_record;
		func_record new_record = {CUDNN_RNN_INF_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);
	}
	else {
		hipdnnStatus_t (*function)(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes);

		*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnRNNForwardInference");
		assert(function != NULL);

		status = (*function)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes);


		printf("------------------------- cudnn status is %d\n", status);
		// TODO: not sure why this complains here in just one call!
		//assert (status == HIPDNN_STATUS_SUCCESS);

		hipError_t err_all = hipDeviceSynchronize(); // for debugging
		CHECK_CUDA_ERROR(err_all);
	}

	return status;

}

hipdnnStatus_t hipdnnDestroyRNNDescriptor(hipdnnRNNDescriptor_t rnnDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyRNNDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyTensorDescriptor(hipdnnTensorDescriptor_t tensorDesc) {

	// mock cudnn destroy TensorDescriptor
	//DEBUG_PRINT("Caught a hipdnnDestroyTensorDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}


hipdnnStatus_t hipdnnDestroyFilterDescriptor(hipdnnFilterDescriptor_t filterDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyFilterDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}


hipdnnStatus_t hipdnnDestroyConvolutionDescriptor(hipdnnConvolutionDescriptor_t convDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyConvolutionDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc) {
	//DEBUG_PRINT("Caught a hipdnnDestroyDropoutDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}
