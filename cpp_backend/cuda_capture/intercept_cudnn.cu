
#include "intercept_temp.h"

void getDescriptor(const hipdnnTensorDescriptor_t desc) {

	int ndims = 10;
	hipdnnDataType_t* dtype = (hipdnnDataType_t*)malloc(sizeof(hipdnnDataType_t));
	int* nbdims = (int*)malloc(sizeof(int));
	int dimA[10] = {0};
	int strideA[10] = {0};

	hipdnnStatus_t status = hipdnnGetTensorNdDescriptor(desc, ndims, dtype, nbdims, dimA, strideA);
	printf("%d\n", *dtype);

	assert (status==HIPDNN_STATUS_SUCCESS);

}

hipdnnStatus_t hipdnnConvolutionForward(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) {


	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH-%d]-[%d] Caught hipdnnConvolutionForward, CUDNN handle is %p\n", idx, func_indexes[idx], handle, idx);

	if (idx < 2)
		block(idx,  mutexes, kqueues);

	// create record
	cudnnConvolutionForward_record new_conv_record = {
		handle,
		alpha,
		xDesc,
		x,
		wDesc,
		w,
		convDesc,
		algo,
		workSpace,
		workSpaceSizeInBytes,
		beta,
		yDesc,
		y
	};
	union func_data new_func_data;
	new_func_data.cudnnConvRecord = new_conv_record;
	func_record new_record = {CUDNN_CONV_RECORD, new_func_data};

	// push or run
	if (idx < 2) {
		 pthread_mutex_lock(mutexes[idx]);
		 kqueues[idx]->push(new_record);
		 pthread_mutex_unlock(mutexes[idx]);

		 func_indexes[idx] += 1;
		 block(idx, mutexes, kqueues);
	}
	else {

		hipdnnStatus_t (*function)(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) ;
		*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnConvolutionForward");
		assert(function != NULL);

		status = (*function)(handle, alpha, xDesc, x, wDesc, w, convDesc, algo, workSpace, workSpaceSizeInBytes, beta, yDesc, y);
		if (status != HIPDNN_STATUS_SUCCESS)
			printf("status is %d\n", status);
		assert (status == HIPDNN_STATUS_SUCCESS);

		DEBUG_PRINT("CONV submitted!!\n");
	}

	return status;

}

hipdnnStatus_t cudnnBatchNormalizationForwardTrainingEx(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH-%d]-[%d] Caught cudnnBatchNormalizationForwardTrainingEx, handle is %p\n", idx, func_indexes[idx], handle);

	if (idx < 2)
		block(idx,  mutexes, kqueues);

	// create record
	cudnnBatchNormalizationForwardTrainingEx_record new_bn_record = {
		handle,
		mode,
		bnOps,
		alpha,
		beta,
		xDesc,
		xData,
		zDesc,
		zData,
		yDesc,
		yData,
		bnScaleBiasMeanVarDesc,
		bnScaleData,
		bnBiasData,
		exponentialAverageFactor,
		resultRunningMeanData,
		resultRunningVarianceData,
		epsilon,
		saveMean,
		saveInvVariance,
		activationDesc,
		workspace,
		workSpaceSizeInBytes,
		reserveSpace,
		reserveSpaceSizeInBytes

	};
	union func_data new_func_data;
	new_func_data.cudnnBNormRecord = new_bn_record;
	func_record new_record = {CUDNN_BNORM_RECORD, new_func_data};

	// push or run

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);
	}
	else {
		hipdnnStatus_t (*function)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes);

		*(void **)(&function) = dlsym(RTLD_NEXT, "cudnnBatchNormalizationForwardTrainingEx");
		assert(function != NULL);

		status = (*function)(handle, mode, bnOps, alpha, beta, xDesc, xData, zDesc, zData, yDesc, yData, bnScaleBiasMeanVarDesc, bnScaleData, bnBiasData, exponentialAverageFactor, resultRunningMeanData, resultRunningVarianceData, epsilon, saveMean, saveInvVariance, activationDesc, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);
		if (status != HIPDNN_STATUS_SUCCESS)
			printf("status is %d\n", status);
		assert (status == HIPDNN_STATUS_SUCCESS);

		DEBUG_PRINT("BNORM submitted!!\n");

	}

	return status;
}


hipdnnStatus_t hipdnnBatchNormalizationForwardInference(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon)

{

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnBatchNormalizationForwardInference, handle is %p, index is %d\n", func_indexes[idx], handle, idx);

	if (idx < 2)
		block(idx,  mutexes, kqueues);


	// create record
	cudnnBatchNormalizationForwardInference_record bn_record = {
		handle,
		mode,
		alpha,
		beta,
		xDesc,
		x,
		yDesc,
		y,
		bnScaleBiasMeanVarDesc,
		bnScale,
		bnBias,
		estimatedMean,
		estimatedVariance,
		epsilon
	};

	union func_data new_func_data;
	new_func_data.cudnnBNormInfRecord = bn_record;
	func_record new_record = {CUDNN_BNORM_INF_RECORD, new_func_data};

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);

	}
	else {

		hipdnnStatus_t (*function)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon);

		*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnBatchNormalizationForwardInference");
		assert(function != NULL);

		status = (*function)(handle, mode, alpha, beta, xDesc, x, xDesc, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, estimatedMean, estimatedVariance, epsilon);
		assert (status == HIPDNN_STATUS_SUCCESS);

	}

	return status;
}


hipdnnStatus_t hipdnnRNNForwardInference(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes)  {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;


	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnRNNForwardInference, handle is %p, index is %d\n", func_indexes[idx], handle, idx);
	printf("------------------------------------------------- IDX [%d], CX IS %p, CY IS %p\n", idx, cx, cy);

	if (idx < 2) {

		hipdnnTensorDescriptor_t* xDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
	        //hipdnnStatus_t s = hipdnnCreateTensorDescriptor(xDesc_new);

		*xDesc_new = *xDesc;
		//printf("%p, %p, %p, %p\n", xDesc, *xDesc, xDesc_new, *(xDesc_new));
		//memcpy(xDesc_new, xDesc, sizeof(hipdnnTensorDescriptor_t));

		hipdnnTensorDescriptor_t* yDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
		*yDesc_new = *yDesc;


		cudnnRNNForwardInf_record rnn_record = {
			handle,
			rnnDesc,
			seqLength,
			xDesc_new,
			x,
			hxDesc,
			hx,
			cxDesc,
			cx,
			wDesc,
			w,
			yDesc_new,
			y,
			hyDesc,
			hy,
			cyDesc,
			cy,
			workspace,
			workSpaceSizeInBytes
		};

		union func_data new_func_data;
		new_func_data.cudnnRnnInfRecord = rnn_record;
		func_record new_record = {CUDNN_RNN_INF_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);
	}
	else {
		hipdnnStatus_t (*function)(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes);

		*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnRNNForwardInference");
		assert(function != NULL);

		status = (*function)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes);


		printf("------------------------- cudnn status is %d\n", status);
		// TODO: not sure why this complains here in just one call!
		assert (status == HIPDNN_STATUS_SUCCESS);

		// hipError_t err_all = hipDeviceSynchronize(); // for debugging
		// CHECK_CUDA_ERROR(err_all);
	}

	return status;

}


hipdnnStatus_t hipdnnRNNForwardTraining(
	hipdnnHandle_t handle,
	const hipdnnRNNDescriptor_t rnnDesc,
	const int seqLength,
	const hipdnnTensorDescriptor_t *xDesc,
	const void *x,
    const hipdnnTensorDescriptor_t hxDesc,
    const void *hx,
    const hipdnnTensorDescriptor_t cxDesc,
    const void *cx,
    const hipdnnFilterDescriptor_t wDesc,
    const void *w,
    const hipdnnTensorDescriptor_t *yDesc,
    void *y,
    const hipdnnTensorDescriptor_t hyDesc,
    void *hy,
    const hipdnnTensorDescriptor_t cyDesc,
    void *cy,
    void *workspace,
    size_t workSpaceSizeInBytes,
    void *reserveSpace,
    size_t reserveSpaceSizeInBytes
) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnRNNForwardTraining, handle is %p, index is %d\n", func_indexes[idx], handle, idx);

	if (idx < 2) {

		hipdnnTensorDescriptor_t* xDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
	        //hipdnnStatus_t s = hipdnnCreateTensorDescriptor(xDesc_new);

		*xDesc_new = *xDesc;
		printf("%p, %p, %p, %p\n", xDesc, *xDesc, xDesc_new, *(xDesc_new));
		//memcpy(xDesc_new, xDesc, sizeof(hipdnnTensorDescriptor_t));

		hipdnnTensorDescriptor_t* yDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
		*yDesc_new = *yDesc;

		cudnnRNNForwardTraining_record rnn_record = {
			handle,
			rnnDesc,
			seqLength,
			xDesc,
			x,
			hxDesc,
			hx,
			cxDesc,
			cx,
			wDesc,
			w,
			yDesc,
			y,
			hyDesc,
			hy,
			cyDesc,
			cy,
			workspace,
			workSpaceSizeInBytes,
			reserveSpace,
			reserveSpaceSizeInBytes
		};

		union func_data new_func_data;
		new_func_data.cudnnRnnTrainRecord = rnn_record;
		func_record new_record = {CUDNN_RNN_TRAIN_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);
	}
	else {

		hipdnnStatus_t (*function)(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes);
		*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnRNNForwardTraining");
		assert(function != NULL);

		status = (*function)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);

		// hipError_t err_all = hipDeviceSynchronize(); // for debugging
		// CHECK_CUDA_ERROR(err_all);
	}

	return status;
}

// hipdnnStatus_t cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
// 	hipdnnHandle_t handle,
// 	hipdnnBatchNormMode_t mode,
// 	cudnnBatchNormOps_t bnOps,
// 	const hipdnnActivationDescriptor_t activationDesc,
// 	const hipdnnTensorDescriptor_t xDesc,
//     size_t *sizeInBytes
// ) {

// 	int idx = get_idx();
// 	assert (idx >= 0);
// 	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

// 	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught cudnnGetBatchNormalizationTrainingExReserveSpaceSize, handle is %p, index is %d\n", func_indexes[idx], handle, idx);


// 	if (idx < 2) {

// 		cudnnBatchNormReserve_record cr_record = {handle, mode, bnOps, activationDesc, xDesc, sizeInBytes};
// 		union func_data new_func_data;
// 		new_func_data.cudnnBNormResRecord = cr_record;
// 		func_record new_record = {CUDNN_BNORM_RESERVE_RECORD, new_func_data};

// 		pthread_mutex_lock(mutexes[idx]);
// 		kqueues[idx]->push(new_record);
// 		pthread_mutex_unlock(mutexes[idx]);

// 		block(idx,  mutexes, kqueues);
// 	}
// 	else {
// 		hipdnnStatus_t (*function)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const hipdnnActivationDescriptor_t activationDesc, const hipdnnTensorDescriptor_t xDesc, size_t *sizeInBytes);
// 		*(void **)(&function) = dlsym(RTLD_NEXT, "cudnnGetBatchNormalizationTrainingExReserveSpaceSize");
// 		assert(function != NULL);

// 		status = (*function)(handle, mode, bnOps, activationDesc, xDesc, sizeInBytes);
// 		if (status != HIPDNN_STATUS_SUCCESS)
// 			printf("status is %d\n", status);
// 		assert (status == HIPDNN_STATUS_SUCCESS);

// 		//CHECK_CUDA_ERROR(hipDeviceSynchronize());
// 	}

// 	return status;

// }

hipdnnStatus_t hipdnnDestroyRNNDescriptor(hipdnnRNNDescriptor_t rnnDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyRNNDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyTensorDescriptor(hipdnnTensorDescriptor_t tensorDesc) {

	// mock cudnn destroy TensorDescriptor
	//DEBUG_PRINT("Caught a hipdnnDestroyTensorDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}


hipdnnStatus_t hipdnnDestroyFilterDescriptor(hipdnnFilterDescriptor_t filterDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyFilterDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}


hipdnnStatus_t hipdnnDestroyConvolutionDescriptor(hipdnnConvolutionDescriptor_t convDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyConvolutionDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc) {
	//DEBUG_PRINT("Caught a hipdnnDestroyDropoutDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}



// hipdnnStatus_t hipdnnCreate(hipdnnHandle_t *handle) {

// 	int idx = get_idx();
// 	assert (idx >= 0);
// 	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

// 	printf("Caught a hipdnnCreate!\n");

// 	// cudnnCreate_record cr_record = {handle};
// 	// union func_data new_func_data;
// 	// new_func_data.cudnnCreateRecord = cr_record;
// 	// func_record new_record = {CUDNN_CREATE_RECORD, new_func_data};

// 	hipdnnStatus_t (*function)(hipdnnHandle_t *handle);
// 	*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnCreate");
// 	assert(function != NULL);

// 	status = (*function)(handle);
// 	assert (status == HIPDNN_STATUS_SUCCESS);

// 	status = hipdnnSetStream(*handle, client_streams[idx]);
// 	if (status != HIPDNN_STATUS_SUCCESS)
// 		printf("status is %d\n", status);
// 	assert (status == HIPDNN_STATUS_SUCCESS);

// 	// if (idx < 2) {

// 	// 	pthread_mutex_lock(mutexes[idx]);
// 	// 	kqueues[idx]->push(new_record);
// 	// 	pthread_mutex_unlock(mutexes[idx]);

// 	// 	block(idx,  mutexes, kqueues);
// 	// }
// 	// else {
// 	// 	hipdnnStatus_t (*function)(hipdnnHandle_t *handle);
// 	// 	*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnCreate");
// 	// 	assert(function != NULL);

// 	// 	status = (*function)(handle);
// 	// 	assert (status == HIPDNN_STATUS_SUCCESS);
// 	// }

// 	return status;
// }


// hipdnnStatus_t hipdnnSetStream(hipdnnHandle_t handle, hipStream_t streamId) {
// 	int idx = get_idx();
// 	assert (idx >= 0);

// 	//printf("Caught a hipdnnSetStream from idx %d, stream is %d!\n", idx, streamId);

// 	if (idx < 2)
// 		return HIPDNN_STATUS_SUCCESS;

// 	hipdnnStatus_t (*function)(hipdnnHandle_t handle, hipStream_t streamId);
// 	*(void **)(&function) = dlsym(RTLD_NEXT, "hipdnnSetStream");
// 	assert(function != NULL);

// 	hipdnnStatus_t status = (*function)(handle, streamId);
// 	//hipError_t err_all = hipDeviceSynchronize(); // for debugging
// 	//CHECK_CUDA_ERROR(err_all);
// 	return status;
// }


hipdnnStatus_t hipdnnDestroy(hipdnnHandle_t handle) {

	printf("Caught a hipdnnDestroy, Do nothing!\n ");
	return HIPDNN_STATUS_SUCCESS;
}